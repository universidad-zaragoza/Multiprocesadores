#define N 1024*1024*1024 


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, long int n) {
    for(long int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
	hipError_t cudaerr;

    printf("%d\n", sizeof(long int));

    // Allocate memory in host memory (CPU)
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(long int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

	// Allocate memory in device memory (GPU)
	float *d_a, *d_b, *d_out;
    cudaerr = hipMalloc((void **) &d_a, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_b, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	cudaerr = hipMalloc((void **) &d_out, sizeof(float)*N);
    if (cudaerr != hipSuccess)
		printf("hipMalloc failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	// Copy source data from host to device
	cudaerr = hipMemcpy (a, d_a, sizeof(float)*N, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess)
		printf("Copying data host to device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
	
	cudaerr = hipMemcpy (b, d_b, sizeof(float)*N, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess)
		printf("Copying data host to device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    
	// Main function
    vector_add<<<1,1>>>(out, a, b, N);

	// Copy result data from device to host 
	cudaerr = hipMemcpy (out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess)
		printf("Copying data Device to host failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	//Free memory in the device	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);


}